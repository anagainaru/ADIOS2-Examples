#include "hip/hip_runtime.h"
#include "cudaRoutines.h"

__global__ void __cuda_increment(float *vec, float val)
{
    vec[blockIdx.x] += val;
}

void cuda_increment(int M, int N, float *vec, float val)
{
    __cuda_increment<<<M, N>>>(vec, val);
}

__global__ void __cuda_initialize(float *vec) { vec[blockIdx.x] = blockIdx.x; }

void cuda_initialize(int M, int N, float *vec)
{
    __cuda_initialize<<<M, N>>>(vec);
}
